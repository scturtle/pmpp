#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define FILTER_RADIUS 1
#define FILTER_DIM (2 * FILTER_RADIUS + 1)

// declare filter F in constant memory
__constant__ float F[FILTER_DIM][FILTER_DIM];

__global__ void convolution_2D_constant_kernel(float *N, float *P, int width, int height) {
  int outCol = blockIdx.x*blockDim.x + threadIdx.x;
  int outRow = blockIdx.y*blockDim.y + threadIdx.y;
  if (outRow < height && outCol < width) {
    float Pvalue = 0.0f;
    for (int fRow = 0; fRow < FILTER_DIM; fRow++) {
      for (int fCol = 0; fCol < FILTER_DIM; fCol++) {
        int inRow = outRow - FILTER_RADIUS + fRow;
        int inCol = outCol - FILTER_RADIUS + fCol;
        if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width) {
          Pvalue += F[fRow][fCol] * N[inRow*width + inCol];
        }
      }
    }
    P[outRow*width + outCol] = Pvalue;
  }
}

void convolution_2D_cpu(const float *h_N, const float *h_F, float *P, int r, int width, int height) {
  int filter_dim = 2 * r + 1;
  for (int outRow = 0; outRow < height; ++outRow) {
    for (int outCol = 0; outCol < width; ++outCol) {
      float Pvalue = 0.0f;
      for (int fRow = 0; fRow < filter_dim; ++fRow) {
        for (int fCol = 0; fCol < filter_dim; ++fCol) {
          int inRow = outRow - r + fRow;
          int inCol = outCol - r + fCol;
          if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width) {
            Pvalue += h_F[fRow * filter_dim + fCol] * h_N[inRow * width + inCol];
          }
        }
      }
      P[outRow * width + outCol] = Pvalue;
    }
  }
}

void verify_results(const float* cpu_result, const float* gpu_result, int size) {
  const float epsilon = 1e-5f;
  int errors = 0;
  for (int i = 0; i < size; ++i) {
    if (fabs(cpu_result[i] - gpu_result[i]) > epsilon) {
      errors++;
    }
  }
  if (errors == 0) {
    printf("Verification successful!\n");
  } else {
    printf("Verification failed with %d errors.\n", errors);
  }
}

int main() {
  int width = 256;
  int height = 256;
  int r = FILTER_RADIUS;
  int filter_dim = FILTER_DIM;

  int input_size = width * height;
  int filter_size = filter_dim * filter_dim;
  int output_size = width * height;

  size_t input_bytes = input_size * sizeof(float);
  size_t filter_bytes = filter_size * sizeof(float);
  size_t output_bytes = output_size * sizeof(float);

  float *h_N, *h_F, *h_P_cpu, *h_P_gpu;
  float *d_N, *d_P;

  h_N = (float*)malloc(input_bytes);
  h_F = (float*)malloc(filter_bytes);
  h_P_cpu = (float*)malloc(output_bytes);
  h_P_gpu = (float*)malloc(output_bytes);

  for (int i = 0; i < input_size; ++i) {
    h_N[i] = (float)(rand() % 10);
  }
  for (int i = 0; i < filter_size; ++i) {
    h_F[i] = (float)(rand() % 3);
  }

  hipMalloc((void**)&d_N, input_bytes);
  hipMalloc((void**)&d_P, output_bytes);

  hipMemcpy(d_N, h_N, input_bytes, hipMemcpyHostToDevice);

  // copy host filter h_F to __constant__ memory symbol F
  hipMemcpyToSymbol(HIP_SYMBOL(F), h_F, filter_bytes);

  dim3 dimBlock(16, 16);
  dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

  convolution_2D_constant_kernel<<<dimGrid, dimBlock>>>(d_N, d_P, width, height);
  hipDeviceSynchronize();

  hipMemcpy(h_P_gpu, d_P, output_bytes, hipMemcpyDeviceToHost);

  convolution_2D_cpu(h_N, h_F, h_P_cpu, r, width, height);

  verify_results(h_P_cpu, h_P_gpu, output_size);

  hipFree(d_N);
  hipFree(d_P);

  free(h_N);
  free(h_F);
  free(h_P_cpu);
  free(h_P_gpu);

  return 0;
}
